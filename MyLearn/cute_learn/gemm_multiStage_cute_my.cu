#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cute/tensor.hpp>

#include "cublaslt-gemm.h"
#include "data.h"


template <typename Config>
__global__ void /* __launch_bounds__(128, 1) */
gemm_multi_stage(void *Dptr, const void *Aptr, const void *Bptr, int m, int n,
                 int k) {
    using namespace cute;

    using T = typename Config::T;
    using SmemLayoutA = typename Config::SmemLayoutA;
    using SmemLayoutB = typename Config::SmemLayoutB;
    using SmemLayoutC = typename Config::SmemLayoutC;
    using TiledMMA = typename Config::MMA;

    using S2RCopyAtomA = typename Config::S2RCopyAtomA;
    using S2RCopyAtomB = typename Config::S2RCopyAtomB;
    using G2SCopyA = typename Config::G2SCopyA;
    using G2SCopyB = typename Config::G2SCopyB;
    using R2SCopyAtomC = typename Config::R2SCopyAtomC;
    using S2GCopyAtomC = typename Config::S2GCopyAtomC;
    using S2GCopyC = typename Config::S2GCopyC;

    constexpr int kTileM = Config::kTileM;
    constexpr int kTileN = Config::kTileN;
    constexpr int kTileK = Config::kTileK;
    constexpr int kStage = Config::kStage;

    extern __shared__ T shm_data[];
    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    Tensor A = make_tensor(make_gmem_ptr((T *)Aptr), make_shape(m, k),
                         make_stride(k, Int<1>{}));  // (M, K)
    Tensor B = make_tensor(make_gmem_ptr((T *)Bptr), make_shape(n, k),
                            make_stride(k, Int<1>{}));  // (N, K)
    Tensor D = make_tensor(make_gmem_ptr((T *)Dptr), make_shape(m, n),
                            make_stride(n, Int<1>{}));  // (M, N)
    // slice the tensor to small one which is used for current thread block.
    Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}),
                            make_coord(iy, _));  // (kTileM, kTileK, k)
    Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}),
                            make_coord(ix, _));  // (kTileN, kTileK, k)
    Tensor gD = local_tile(D, make_tile(Int<kTileM>{}, Int<kTileN>{}),
                            make_coord(iy, ix));  // (kTileM, kTileN)
    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm),
                            SmemLayoutA{});  // (kTileM, kTileK, kStage)
    auto sB = make_tensor(make_smem_ptr(Bshm),
                            SmemLayoutB{});  // (kTileN, kTileK, kStage)
    
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(idx);
    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);           // (MMA, MMA_M, MMA_N)
    clear(tCrD);

    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA);  // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA);  // (CPY, CPY_M, CPY_K, kStage)
    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB);  // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB);  // (CPY, CPY_N, CPY_K, kStage)

    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);  // ? (CPY, CPY_M, CPY_K, kStage)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA);  // ? (CPY, CPY_M, CPY_K)
    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tBsB = s2r_thr_copy_b.partition_S(sB);  // ? (CPY, CPY_N, CPY_K, kStage)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB);  // ? (CPY, CPY_N, CPY_K)

    int itile_to_read = 0;
    int ismem_read = 0;
    int ismem_write = 0;
    // submit kStage-1 tile
    for(int istage = 0; istage < kStage-1; istage++){
        cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, istage), tAsA_copy(_, _, _, istage));
        cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, istage), tBsB_copy(_, _, _, istage));
        cp_async_fence();       // commit
        itile_to_read++;
        ismem_write++;
    }
    // wait one submitted gmem->smem done
    cp_async_wait<kStage - 2>();        // 这里发射了kStage-1个tile的读，等至少一个结束，那么放过kStage-2个
    __syncthreads();

    int ik = 0;
    // smem -> reg
    cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik, ismem_read), tCrA_view(_, _, ik));
    cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik, ismem_read), tCrB_view(_, _, ik));

    // main loop
    int ntile = k / kTileK;
    #pragma unroll  1
    for(int itile = 0; itile < ntile; itile++){
        int nk = size<2>(tCrA);     // MMA_K
        #pragma unroll
        for(ik = 0; ik < nk; ik++){
            int ik_next = (ik+1) % nk;

            if (ik == nk - 1) {
                cp_async_wait<kStage - 2>();
                __syncthreads();
                ismem_read = (ismem_read + 1) % kStage;
            }

            // shm -> reg s[itile][ik + 1] -> r[ik + 1]
            cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik_next, ismem_read),
                        tCrA_view(_, _, ik_next));
            cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik_next, ismem_read),
                        tCrB_view(_, _, ik_next));
            
            if(ik == 0){
                if(tile_to_read < ntile){
                    cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile_to_read), tAsA_copy(_, _, _, ismem_write));
                    cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile_to_read), tBsB_copy(_, _, _, ismem_write));
                    itile_to_read++;
                    ismem_write = (ismem_write+1) % kStage;
                }
                cp_async_fence();       // commit
            }

            cute::gemm(tiled_mma, tCrD, tCrA, tCrB, tCrD);
        }
    }
    // epilogue
    auto sC = make_tensor(sA(_, _, ismem_read).data(), SmemLayoutC{});
    auto r2s_tiled_copy_c = make_tiled_copy_C(R2SCopyAtomC{}, tiled_mma);
    auto r2s_thr_copy_c = r2s_tiled_copy_c.get_slice(idx);
    auto tCrC_r2s = r2s_thr_copy_c.retile_S(tCrD);   // (CPY, CPY_M, CPY_N)
    auto tCsC_r2s = r2s_thr_copy_c.partition_D(sC);  // (CPY, _1, _1, pipe)
    S2GCopyC s2g_tiled_copy_c;
    auto s2g_thr_copy_c = s2g_tiled_copy_c.get_thread_slice(idx);
    auto tCsC_s2g = s2g_thr_copy_c.partition_S(sC);  // (CPY, _1, _1, pipe)
    auto tCgC_s2g = s2g_thr_copy_c.partition_D(gD);  // (CPY, CPY_M, CPY_N)

    auto tCgC_s2gx = group_modes<1, 3>(tCgC_s2g);  // (CPY_, CPY_MN)
    auto tCrC_r2sx = group_modes<1, 3>(tCrC_r2s);  // (CPY_, CPY_MN)

    int step = size<3>(tCsC_r2s);  // pipe
    #pragma unroll
    for(int i = 0; i < size<1>(tCrC_r2sx); i+=step){
        #pragma unroll
        for(int j = 0; j < step; j++){
            auto t = make_tensor_like<T>(tCrC_r2sx(_, i+j));
            cute::copy(tCrC_r2sx(_, i+j), t);
            cute::copy(r2s_tiled_copy_c, t, tCsC_r2s(_, 0, 0, j));
        }
        __syncthreads();
        #pragma unroll
        for(int j = 0; j < step; j++){
            cute::copy(s2g_tiled_copy_c, tCsC_s2g(_, 0, 0, j), tCgC_s2gx(_, i+j));
        }
        __syncthreads();
    }
}

template<typename T_, int kTileM_=128, int kTileN_=128, int kTileK_=32,
         int kStage_=5, int kSmemLayoutCBatch_=2, typename ComputeType = T_>
struct GemmConfig{
    using T = T_;
    static constexpr int kTileM = kTileM_;
    static constexpr int kTileN = kTileN_;
    static constexpr int kTileK = kTileK_;
    static constexpr int kStage = kStage_;
    static constexpr int kSmemLayoutCBatch = kSmemLayoutCBatch_;

    static constexpr int kShmLoadSwizzleM = 3;
    static constexpr int kShmLoadSwizzleS = 3;
    static constexpr int kShmLoadSwizzleB = 3;
    // 8*32个half Swizzle成 4*(8*8)。irow的更新间隔为8。
    using SmemLayoutAtom = decltype(composition(
        Swizzle<kShmLoadSwizzleB, kShmLoadSwizzleM, kShmLoadSwizzleS>{},
        make_layout(make_shape(Int<8>{}, Int<kTileK>{}),
                    make_stride(Int<kTileK>{}, Int<1>{}))
    ));
    using SmemLayoutA = decltype(
      tile_to_shape(SmemLayoutAtom{},
                    make_shape(Int<kTileM>{}, Int<kTileK>{}, Int<kStage>{})));
    using SmemLayoutB = decltype(
      tile_to_shape(SmemLayoutAtom{},
                    make_shape(Int<kTileN>{}, Int<kTileK>{}, Int<kStage>{})));

    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;

    static constexpr int kMmaEURepeatM = 2;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});

    using MMA_EU_RepeatT = decltype(make_layout(
        make_shape(Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));

    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>; 

    using G2SCopyA = decltype(make_tiled_copy(g2s_copy_atom{},
                                              make_layout(make_shape(Int<32>{}, Int<4>{}),
                                                          make_stride(Int<4>{}, Int<1>{})),
                                              make_layout(make_shape(Int<1>{}, Int<8>{}))));
    using G2SCopyB = G2SCopyA;

    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;

    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    /// epilogue
    using SmemLayoutAtomC = decltype(composition(
        Swizzle<2,3,3>{}, make_layout(make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}),
                                      make_stride(Int<kMmaPN>{}, Int<1>{}))
    ));
    using SmemLayoutC = decltype(tile_to_shape(
        SmemLayoutAtomC{},
        make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}, Int<kSmemLayoutCBatch>{})));

    static_assert(size<0>(SmemLayoutA{}) * size<1>(SmemLayoutA{}) >=
                    size(SmemLayoutC{}),
                "C shared memory request is large than A's one pipe");

    using R2SCopyAtomC = Copy_Atom<UniversalCopy<int>, T>;
    using S2GCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
    using S2GCopyC =
        decltype(make_tiled_copy(S2GCopyAtomC{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}),
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{}))));

    static constexpr int kThreadNum = size(MMA{});
    static constexpr int shm_size_AB = cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
    static constexpr int shm_size_C = cute::cosize(SmemLayoutC{}); 
    static constexpr int kShmSize = cute::max(shm_size_AB, shm_size_C) * sizeof(T);
}

int main(){
    using T = cute::half_t;
    using namespace cute;
    int M = 81920;
    int N = 256;
    int K = 256;

    GemmConfig<T, 128, 128, 32, 3> gemm_config;

    dim3 block = gemm_config.kThreadNum;
    dim3 grid((N + gemm_config.kTileN - 1) / gemm_config.kTileN,
            (M + gemm_config.kTileM - 1) / gemm_config.kTileM);
    int shm_size = gemm_config.kShmSize;
    gemm_multi_stage<decltype(gemm_config)>
        <<<grid, block, shm_size>>>(Dptr, Aptr, Bptr, M, N, K);
}
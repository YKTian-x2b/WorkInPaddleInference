#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>  
  
// 核函数，用于生成随机数  
__global__ void generateRandomNumbers(float* output, hiprandState* state, int N) {  
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  
    if (tid < N) {  
        hiprandState localState = state[tid];  
        output[tid] = hiprand_uniform(&localState);  
        state[tid] = localState; // 更新状态（如果需要）  
    }  
}  
  
int main() {  
    // 初始化变量和内存  
    const int N = 1024; // 要生成的随机数的数量  
    float* h_output; // 主机上的输出数组  
    hiprandState* d_state; // 设备上的状态数组  
    float* d_output; // 设备上的输出数组  
  
    // 分配主机内存  
    h_output = (float*)malloc(N * sizeof(float));  
  
    // 分配设备内存  
    hipMalloc((void**)&d_output, N * sizeof(float));  
    hipMalloc((void**)&d_state, N * sizeof(hiprandState));  
  
    // 初始化随机数生成器的状态（在主机上）  
    hiprandState* h_state = (hiprandState*)malloc(N * sizeof(hiprandState));  
    for (int i = 0; i < N; ++i) {  
        hiprand_init(time(0), i, 0, &h_state[i]); // 使用当前时间和索引作为种子  
    }  
  
    // 将状态从主机复制到设备  
    hipMemcpy(d_state, h_state, N * sizeof(hiprandState), hipMemcpyHostToDevice);  
  
    // 调用核函数生成随机数  
    generateRandomNumbers<<<1, N>>>(d_output, d_state, N);  
  
    // 将输出从设备复制回主机  
    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);  
  
    // 清理  
    free(h_output);  
    free(h_state);  
    hipFree(d_output);  
    hipFree(d_state);  
  
    return 0;  
}